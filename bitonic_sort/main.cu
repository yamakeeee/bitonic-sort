#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>

#include "hip/hip_runtime.h"

static void bitonic_sort_GPU(int *in, int*out, const int n);
// static void bitonic_sort_CPU(int *in, int*out, const int n);

int main()
{
    printf("Hello from CPU\n");

    const int n = 1024;
    int *in = new int[n];
    int *out_cpu = new int[n];
    int *out_gpu = new int[n];

    for (int i = 0; i < n; i++) in[i] = rand() % 10000;

    bitonic_sort_GPU(in, out_gpu, n);
    // bitonic_sort_CPU(in, out_cpu, n);

    delete[] in;
    delete[] out_cpu;
    delete[] out_gpu;

    return 0;
}

__global__ void kernel_bitonic_sort(int *in, int *out, const int n)
{
    int i = blockIdx.x * blockDim.x +threadIdx.x;
    if (i < n) {
        out[i] = in[i] * 2;
    }
}

static void bitonic_sort_GPU(int *hIn, int *hOut, const int n)
{
    int *dIn;
    int *dOut;
    hipHostMalloc((void**)&dIn, n * sizeof(int));
    hipHostMalloc((void**)&dOut, n * sizeof(int));
    hipMemcpy(dIn, hIn, n * sizeof(int), hipMemcpyHostToDevice);
    int m=(n+31)/32;
    kernel_vecDouble<<<32, m>>>(dIn, dOut, n);
    hipDeviceSynchronize();

    hipMemcpy(hOut, dOut, n * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dIn);
    hipFree(dOut);
}

// static void bitonic_sort_CPU(int *hIn, int *hOut, const int n){
//     return;
// }
