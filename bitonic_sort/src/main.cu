#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <algorithm>

#include "hip/hip_runtime.h"

static void bitonic_sort_GPU(int *in, int*out, const int n, const int m);
static void bitonic_sort_CPU(int *in, int*out, const int n, const int m);

bool solve()
{
    const int m = 10;
    const int n = (1 << m);

    int *a = new int[n];
    int *out_cpu = new int[n];
    int *out_gpu = new int[n];

    for (int i = 0; i < n; i++) a[i] = rand() % 10000;

    bitonic_sort_GPU(a, out_gpu, n, m);
    bitonic_sort_CPU(a, out_cpu, n, m);

    std::sort(a, a + n);

    for(int i = 0; i < n; ++i){
        if(a[i] != out_cpu[i] || a[i] != out_gpu[i]){
            std::cout<<"NG"<<std::endl;
            printf("a[%d], out_cpu[%d], out_gpu[%d] = %d, %d, %d\n", i, i, i, a[i], out_cpu[i], out_gpu[i]);
            return false;
        }
    }

    delete[] a;
    delete[] out_cpu;
    delete[] out_gpu;
    return true;
}

__global__ void kernel_bitonic_sort(int *a, const int n, const int c, const int j)
{
    int i = blockIdx.x * blockDim.x +threadIdx.x;
    if (i < n / 2) {
        int idx = i + ((i >> j) << j);
        if (((i >> c) & 1) == 0 && a[idx] > a[idx + (1 << j)]){
            int tmp = a[idx];
            a[idx] = a[idx + (1 << j)];
            a[idx + (1 << j)] = tmp;
        }
        else if(((i >> c) & 1) && a[idx] < a[idx + (1 << j)]){
            int tmp = a[idx];
            a[idx] = a[idx + (1 << j)];
            a[idx + (1 << j)] = tmp;
        }
    }
}

static void bitonic_sort_GPU(int *hIn, int *hOut, const int n, const int m)
{
    int *dArray;
    hipHostMalloc((void**)&dArray, n * sizeof(int), hipHostMallocDefault);
    hipMemcpy(dArray, hIn, n * sizeof(int), hipMemcpyHostToDevice);
    int blockSize = (n + 31) / 32;
    for(int c = 0; c < m; ++c){
        for(int j = c; j >= 0; --j){
            kernel_bitonic_sort<<<32, blockSize>>>(dArray, n, c, j);
        }
    }
    hipDeviceSynchronize();
    hipMemcpy(hOut, dArray, n * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dArray);
}

static void bitonic_sort_CPU(int *hIn, int *a, const int n, const int m){
    for(int i = 0; i < n; ++i)a[i] = hIn[i];
    for(int c = 0; c < m; ++c){
        for(int j = c; j >= 0; --j){
            for(int i = 0; i < (n >> 1); ++i){
                int idx = i + ((i >> j) << j);
                // printf("idx, idx + (1 << j) = %d, %d\n", idx, idx + (1 << j));
                if (((i >> c) & 1) == 0 && a[idx] > a[idx + (1 << j)]){
                    std::swap(a[idx], a[idx + (1 << j)]);
                }
                else if(((i >> c) & 1) && a[idx] < a[idx + (1 << j)]){
                    std::swap(a[idx], a[idx + (1 << j)]);
                }
            }
        }
    }
    return;
}

int main(){
    int n = 1000;
    int ng = 0;
    for(int i = 0; i < n; ++i){
        if(!solve())++ng;
    }
    std::cout << ng << std::endl;
}